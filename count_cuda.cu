#include <iostream>
#include <fstream>
#include <unordered_map>
#include <string>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>

const int PRIME = 37;
const int MOD = 1e9 + 7;
// cuda kernel for counting substrings using hash
__global__ void countSubstringsKernel(const char* content, int* substringCount, int contentLength, int maxSubstringLength) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= contentLength) return;
    // printf("Block ID: %d, Block Dim: %d, Thread ID: %d\n", blockIdx.x, blockDim.x, threadIdx.x);
    // std::cout<<blockIdx.x<<"and"<<blockDim.x<<"and"<<threadIdx.x;


    for (int len = 1; len <= maxSubstringLength; ++len) {
        int hashValue = 0;
        int power = 1;
        // compute the hash for the current substring
        for (int j = i; j < i + len && j < contentLength; ++j) {
            hashValue = (hashValue + (content[j] - 'a' + 1) * power) % MOD;
            power = (power * PRIME) % MOD;

        }

        // atomically increment the hash count
        atomicAdd(&substringCount[hashValue], 1);
    }
}

std::string readFile(const std::string& filename) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Error opening file!" << std::endl;
        return "0";
    }
    std::string line, content;

    while (getline(file, line)) {
        content += line + " "; // changing multiple lines of text into a single string
    }

    return content;
}

// counting substring frequencies
void countSubstringFrequencies(const std::string& filename, std::unordered_map<int, int>& substringCount, int maxSubstringLength) {
    std::string content = readFile(filename);
    int contentLength = content.length();

    // device memory_allocation
    char* d_content;
    int* d_substringCount;

    hipMalloc(&d_content, contentLength * sizeof(char));
    hipMalloc(&d_substringCount, MOD * sizeof(int));
    // std::cout<<content.c_str();

    hipMemcpy(d_content, content.c_str(), contentLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemset(d_substringCount, 0, MOD * sizeof(int));
    char* h_content = new char[contentLength];

    // defining block size
    int blockSize = 1024;
    int gridSize = (contentLength + blockSize - 1) / blockSize;

    // launching cuda kernel
    countSubstringsKernel<<<gridSize, blockSize>>>(d_content, d_substringCount, contentLength, maxSubstringLength);

    hipDeviceSynchronize();

    // back to host
    int* h_substringCount = new int[MOD];
    hipMemcpy(h_substringCount, d_substringCount, MOD * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < MOD; ++i) {
        if (h_substringCount[i] > 0) {
            substringCount[i] = h_substringCount[i];
        }
    }

    delete[] h_substringCount;
    hipFree(d_content);
    hipFree(d_substringCount);
}

// calculating hash for a substring
int calculateHash(const std::string& str) {
    int hashValue = 0;
    int power = 1;

    for (char c : str) {
        hashValue = (hashValue  + (c - 'a' + 1) * power) % MOD;
        power = (power * PRIME) % MOD;
    }

    return hashValue;
}

// searching substring frequency
void searchSubstringFrequency(const std::unordered_map<int, int>& substringCount, const std::string& substring) {
    int hashValue = calculateHash(substring);
    auto it = substringCount.find(hashValue);
    if (it != substringCount.end()) {
        std::cout << "Frequency of '" << substring << "': " << it->second << std::endl;
    } else {
        std::cout << "Substring not found." << std::endl;
    }
}

int main() {
    // txt file path
    std::string filename = "/mnt/c/Users/abhis/Desktop/RocketGPT/Dataset_files/manual/count.txt";
    auto start_time = std::chrono::high_resolution_clock::now();

    std::unordered_map<int, int> substringCount;

    // maximum length of substring
    int maxSubstringLength = 500;


    // counting all substrings possible
    countSubstringFrequencies(filename, substringCount, maxSubstringLength);
    
    std::string searchTerm = "aerot"; // substring to be serached

    searchSubstringFrequency(substringCount, searchTerm);

    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_time - start_time;
    std::cout << "Simulation completed in " << duration.count() << " seconds." << std::endl;

    return 0;
}